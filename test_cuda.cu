
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void dummy_kernel() { }

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA devices: %d\n", deviceCount);
    
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
    }
    
    dummy_kernel<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}
